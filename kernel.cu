﻿#include <stdio.h>
#include "SDL.h"

#pragma comment(lib, "SDL2main.lib")
#pragma comment(lib, "SDL2.lib")

SDL_Window* window;
SDL_Renderer* renderer;

int SDL_main(int argc, char* argv[])
{
	printf("Start\n");

	// Initialize SDL
	if (SDL_Init(SDL_INIT_VIDEO) < 0) {
		printf("Could not initialize SDL! (%s)\n", SDL_GetError());
		return -1;
	}

	// Create window
	window = SDL_CreateWindow("", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, 640, 480, SDL_WINDOW_OPENGL);
	if (window == NULL) {
		printf("Could not create window! (%s)\n", SDL_GetError());
		return -1;
	}

	// Create renderer
	renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC | SDL_RENDERER_TARGETTEXTURE);
	if (renderer == NULL) {
		printf("Could not create renderer! (%s)\n", SDL_GetError());
		return -1;
	}

	// Clear renderer (white)
	SDL_SetRenderDrawColor(renderer, 255, 255, 255, SDL_ALPHA_OPAQUE);
	SDL_RenderClear(renderer);

	// Draw rect (red)
	SDL_Rect r = { 50, 50, 100, 100 };
	SDL_SetRenderDrawColor(renderer, 255, 0, 0, SDL_ALPHA_OPAQUE);
	SDL_RenderDrawRect(renderer, &r);

	// Update screen
	SDL_RenderPresent(renderer);

	SDL_Event event;
	int done = 0;

	while (!done) {
		SDL_PollEvent(&event);

		if (event.type == SDL_QUIT) {
			done = 1;
		}
	}

	SDL_DestroyRenderer(renderer);
	SDL_DestroyWindow(window);
	SDL_Quit();

	printf("End\n");

	return 0;
}
출처: https://infoarts.tistory.com/45 [방바닥 디자인:티스토리]